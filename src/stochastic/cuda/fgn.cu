#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hipfft/hipfft.h>
#include <hip/hip_complex.h>
#include <math.h>

#ifdef _WIN32
#define EXPORT __declspec(dllexport)
#else
#define EXPORT
#endif

__global__ void fill_random_with_eigs(
    hipComplex* d_data,
    const hipComplex* d_sqrt_eigs,
    int traj_size,
    int m,
    unsigned long seed)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= m * traj_size) return;
    int traj_id = tid / traj_size;
    int idx = tid % traj_size;
    hiprandState state;
    hiprand_init(seed + traj_id, idx, 0, &state);
    float re = hiprand_normal(&state);
    float im = hiprand_normal(&state);
    hipComplex noise = make_hipComplex(re, im);
    d_data[tid] = hipCmulf(noise, d_sqrt_eigs[idx]);
}

__global__ void scale_and_copy_to_output(
    const hipComplex* d_data,
    float* d_output,
    int n,
    int m,
    int offset,
    float hurst,
    float t)
{
    int out_size = n - offset;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= m * out_size) return;
    int traj_id = tid / out_size;
    int idx = tid % out_size;
    int data_idx = traj_id * (2 * n) + (idx + 1);
    float scale = powf((float)n, -hurst) * powf(t, hurst);
    d_output[tid] = d_data[data_idx].x * scale;
}

extern "C" EXPORT void fgn_kernel(
    const hipComplex* d_sqrt_eigs,
    float* d_output,
    int n,
    int m,
    int offset,
    float hurst,
    float t,
    unsigned long seed)
{
    int traj_size = 2 * n;
    hipComplex* d_data = nullptr;
    hipMalloc(&d_data, (size_t)m * traj_size * sizeof(hipComplex));
    {
        int totalThreads = m * traj_size;
        int blockSize = 512;
        int gridSize = (totalThreads + blockSize - 1) / blockSize;
        fill_random_with_eigs<<<gridSize, blockSize>>>(d_data, d_sqrt_eigs, traj_size, m, seed);
        hipDeviceSynchronize();
    }
    {
        hipfftHandle plan;
        hipfftPlan1d(&plan, traj_size, HIPFFT_C2C, m);
        hipfftExecC2C(plan, d_data, d_data, HIPFFT_FORWARD);
        hipDeviceSynchronize();
        hipfftDestroy(plan);
    }
    {
        int out_size = n - offset;
        int totalThreads = m * out_size;
        int blockSize = 512;
        int gridSize = (totalThreads + blockSize - 1) / blockSize;
        scale_and_copy_to_output<<<gridSize, blockSize>>>(d_data, d_output, n, m, offset, hurst, t);
        hipDeviceSynchronize();
    }
    hipFree(d_data);
}
