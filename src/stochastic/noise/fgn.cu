#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <math.h>

extern "C" __global__ void fgn_kernel(
    const hipComplex* sqrt_eigenvalues,
    float* output,
    int n,
    int m,
    int offset,
    float hurst,
    float t,
    unsigned long seed
) {
    int traj_idx = blockIdx.x;
    if (traj_idx >= m) return;

    int idx = threadIdx.x;
    int traj_size = 2 * n;
    int output_size = n - offset;

    extern __shared__ hipComplex shared_data[];

    if (idx < traj_size) {
        hiprandState state;
        hiprand_init(seed + traj_idx, idx, 0, &state);

        float real = hiprand_normal(&state);
        float imag = hiprand_normal(&state);
        hipComplex noise = make_hipComplex(real, imag);
        shared_data[idx] = hipCmulf(noise, sqrt_eigenvalues[idx]);
    }

    __syncthreads();

    if (idx == 0) {
        hipfftHandle plan;
        hipfftComplex* data = (hipfftComplex*)shared_data;
        // TODO: need to optimize, because create a plan for FFT in every thread not efficient
        hipfftPlan1d(&plan, traj_size, HIPFFT_C2C, 1);
        hipfftExecC2C(plan, data, data, HIPFFT_FORWARD);
        hipfftDestroy(plan);
    }

    __syncthreads();

    float scale = powf((float)n, -hurst) * powf(t, hurst);
    if (idx < output_size) {
        int output_offset = traj_idx * output_size;
        output[output_offset + idx] = shared_data[idx + 1].x * scale;
    }
}
