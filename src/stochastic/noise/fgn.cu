#include "hip/hip_runtime.h"
#include "fgn.cuh"
#include <hip/hip_complex.h>
#include <hiprand/hiprand_kernel.h>
#include <hipfft/hipfft.h>
#include <math.h>


__global__ void r_kernel(float* r, int n, float hurst)
{

}

__global__ void sqrt_eigenvalues_kernel(hipfftComplex* r_fft, hipComplex* sqrt_eigenvalues, int n)
{

}

extern "C" void sqrt_eigenvalues_kernel_wrapper(hipComplex* sqrt_eigenvalues, int n, int hurst)
{

}

__global__ void fgn_kernel(const hipComplex* sqrt_eigenvalues, hipComplex* result, int n, float scale, unsigned long seed)
{

}

extern "C" void fgn_kernel_wrapper(const hipComplex* sqrt_eigenvalues, hipComplex* result, int n, int m, float scale, unsigned long seed)
{

}
